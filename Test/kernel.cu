#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>    


// 0+1+2+...+SIZE
#define SIZE 10

__global__ void histo_kernel(int size, unsigned int *histo)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < size)
    {
         //*histo += i;  
        atomicAdd(histo, i);
    }
}

int main(void)
{
    int sum = 0;

    //分配内存并拷贝初始数据  
    unsigned int *dev_histo;
    hipMalloc((void**)&dev_histo, sizeof(int));
    hipMemcpy(dev_histo, &sum, sizeof(int), hipMemcpyHostToDevice);

    // kernel launch - 2x the number of mps gave best timing    
    hipDeviceProp_t  prop;
    hipGetDeviceProperties(&prop, 0);
    int blocks = prop.multiProcessorCount;

    //确保线程数足够  
    histo_kernel <<<blocks * 2, (SIZE + 2 * blocks - 1) / blocks / 2 >>> (SIZE, dev_histo);

    //数据拷贝回CPU内存  
    hipMemcpy(&sum, dev_histo, sizeof(int), hipMemcpyDeviceToHost);
    printf("Threads sum = %d\n", sum);

    hipFree(dev_histo);
    return 0;
}