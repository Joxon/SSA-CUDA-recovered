#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>    


// 0+1+2+...+SIZE
#define SIZE 10

__global__ void histo_kernel(int size, unsigned int *histo)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < size)
    {
         //*histo += i;  
        atomicAdd(histo, i);
    }
}

int main(void)
{
    int sum = 0;

    //�����ڴ沢������ʼ����  
    unsigned int *dev_histo;
    hipMalloc((void**)&dev_histo, sizeof(int));
    hipMemcpy(dev_histo, &sum, sizeof(int), hipMemcpyHostToDevice);

    // kernel launch - 2x the number of mps gave best timing    
    hipDeviceProp_t  prop;
    hipGetDeviceProperties(&prop, 0);
    int blocks = prop.multiProcessorCount;

    //ȷ���߳����㹻  
    histo_kernel <<<blocks * 2, (SIZE + 2 * blocks - 1) / blocks / 2 >>> (SIZE, dev_histo);

    //���ݿ�����CPU�ڴ�  
    hipMemcpy(&sum, dev_histo, sizeof(int), hipMemcpyDeviceToHost);
    printf("Threads sum = %d\n", sum);

    hipFree(dev_histo);
    return 0;
}